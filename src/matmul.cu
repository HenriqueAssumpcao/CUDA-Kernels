#include <hip/driver_types.h>
#include <hip/hip_runtime.h>


#define CUDA_CHECK_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#define CEIL_DIV(x, y) (x + y - 1) / y

// KERNELS
/*
Naive matmul kernel.
Assumes row-major arrays.
A: M x K
B: K x N
C: M x N
*/
__global__ void matmul_naive(const float *A,const float *B,float *C,
                             int M, int K, int N)
{
    size_t trow = threadIdx.y;
    size_t tcol = threadIdx.x;

    size_t row =  blockDim.y*blockIdx.y + trow;
    size_t col = blockDim.x*blockIdx.x + tcol;

    if(row < M && col < N){
        double dot = 0.0;
        for(size_t k = 0; k < K; k++){
            dot += A[row*K + k]*B[k*N + col];
        }
        C[row*N + col] = dot;
    }
}

/*
Global Memory Coalescing matmul kernel.
*/
template <const int WARP_SZ>
__global__ void matmul_coalesce(const float *A,const float *B,float *C,
                                int M, int K, int N)
{
    size_t trow = threadIdx.x / WARP_SZ;
    size_t tcol = threadIdx.x % WARP_SZ;

    size_t row = blockDim.y*blockIdx.y + trow;
    size_t col = blockDim.x*blockIdx.x + tcol;

    if(row < M && col < N){
        double dot = 0.0;
        for(size_t k = 0; k < K; k++){
            dot += A[row*K + k]*B[k*N + col];
        }
        C[row*N + col] = dot;
    }
}

/*
Shared memory and gmc matmul kernel.
Assumes blockDim(WARP_SZ*WARP_SZ,1,1),gridDim(CEIL_DIV(M,WARP_SZ),CEIL_DIV(N,WARP_SZ),1).
*/
template <const int WARP_SZ>
__global__ void matmul_shared(const float *A,const float *B,float *C,
                                int M, int K, int N)
{
    size_t trow = threadIdx.x / WARP_SZ;
    size_t tcol = threadIdx.x % WARP_SZ;

    __shared__ float s_A[WARP_SZ*WARP_SZ],s_B[WARP_SZ*WARP_SZ];

    A += blockIdx.y * WARP_SZ * K;
    B += blockIdx.x * WARP_SZ;
    C += (blockIdx.y * WARP_SZ * N) + (blockIdx.x * WARP_SZ);
    
    double dot = 0.0;

    for(size_t bIdx = 0; bIdx < K; bIdx += WARP_SZ){
        s_A[trow*WARP_SZ + tcol] = A[trow*K + tcol];
        s_B[trow*WARP_SZ + tcol] = B[trow*N + tcol];

        __syncthreads();

        for(size_t k = 0;k < WARP_SZ; ++k){
            dot += s_A[trow*WARP_SZ + k]*s_B[k*WARP_SZ + tcol];
        }

        A += WARP_SZ;
        B += (WARP_SZ*N);

        __syncthreads();   

    }
    C[trow*N + tcol] = dot;
}



// template <const int WARP_SZ>
// __host__ void run_matmul_kernel(const float *h_A,const float *h_B,float *h_C,
//                                 int M, int K, int N,
//                                 void(*matmul_kernel)(const float*, const float*, float*, int, int, int),
//                                 const dim3 gridDim, const dim3 blockDim)
// {
//     // alloc and copy to device
//     float *d_A,*d_B,*d_C;

//     CUDA_CHECK_ERROR(cudaMalloc(&d_A,sizeof(h_A)));
//     CUDA_CHECK_ERROR(cudaMemcpy(d_A,h_A,sizeof(h_A),cudaMemcpyHostToDevice));

//     CUDA_CHECK_ERROR(cudaMalloc(&d_B,sizeof(h_B)));
//     CUDA_CHECK_ERROR(cudaMemcpy(d_B,h_B,sizeof(h_B),cudaMemcpyHostToDevice));

//     CUDA_CHECK_ERROR(cudaMalloc(&d_C,sizeof(h_C)));
//     CUDA_CHECK_ERROR(cudaMemcpy(d_C,h_C,sizeof(h_C),cudaMemcpyHostToDevice));

//     // run kernel
//     matmul_kernel<<<gridDim,blockDim>>>(d_A,d_B,d_C,M,N,K);

//     // back to host
//     CUDA_CHECK_ERROR(cudaMemcpy(h_A,d_A,sizeof(d_A),cudaMemcpyDeviceToHost));
//     CUDA_CHECK_ERROR(cudaMemcpy(h_B,d_B,sizeof(d_B),cudaMemcpyDeviceToHost));
//     CUDA_CHECK_ERROR(cudaMemcpy(h_C,d_C,sizeof(d_C),cudaMemcpyDeviceToHost));

//     // free on device
//     CUDA_CHECK_ERROR(cudaFree(d_A));
//     CUDA_CHECK_ERROR(cudaFree(d_B));
//     CUDA_CHECK_ERROR(cudaFree(d_C));
// }